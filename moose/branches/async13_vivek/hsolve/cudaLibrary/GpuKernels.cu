#include "hip/hip_runtime.h"
/**********************************************************************
** This program is part of 'MOOSE', the
** Messaging Object Oriented Simulation Environment.
**   copyright (C) 2003-2007 Upinder S. Bhalla, Niraj Dudani and NCBS
** It is made available under the terms of the
** GNU Lesser General Public License version 2.1
** See the file COPYING.LIB for the full notice.
**********************************************************************/

#include <cstdio>
#include "GpuKernels.h"
#include "../HSolveStruct.h"	// For CompartmentStruct, etc.
#include "../HinesMatrix.h"		// For JunctionStruct
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/*
 * Check CUDA return value and handle appropriately
 */
#define _(value) {															\
	{																		\
		hipError_t _m_cudaStat = value;									\
		if (_m_cudaStat != hipSuccess) {									\
			std::cerr << "Error " << hipGetErrorString(_m_cudaStat)		\
					  << " at line " << __LINE__ << " in file "				\
					  << __FILE__ << std::endl;								\
			exit(1);														\
		}																	\
	}																		\
}

// Constants needed for active channels
// Note: cuda constant memory is implicitly static. Better to make it
// explicitly known here.
__constant__ static int INSTANT_X;
__constant__ static int INSTANT_Y;
__constant__ static int INSTANT_Z;

void setInstantXYZ(int x, int y, int z)
{
	_( hipMemcpyToSymbol(HIP_SYMBOL( INSTANT_X), &x, sizeof(int) ) );
	_( hipMemcpyToSymbol(HIP_SYMBOL( INSTANT_Y), &y, sizeof(int) ) );
	_( hipMemcpyToSymbol(HIP_SYMBOL( INSTANT_Z), &z, sizeof(int) ) );
}

__global__ void updateMatrixKernel(GpuDataStruct ds) {
	/*
	 * Copy contents of HJCopy_ into HJ_. Cannot do a vector assign() because
	 * iterators to HJ_ get invalidated in MS VC++
	 */
	if ( ds.HJSize != 0 )
		memcpy( ds.HJ, ds.HJCopy, sizeof( double ) * ds.HJSize );

	double *ihs = ds.HS;
	double *iv  = ds.V;
	
	//printf("In gpu: %lf %lf\n", ihs[0], ihs[3]);

	CompartmentStruct *ic;
	for ( ic = ds.compartment ; ic < ds.compartment + ds.nCompts ; ++ic ) {
		*ihs         = *( 2 + ihs );
		*( 3 + ihs ) = *iv * ic->CmByDt + ic->EmByRm;
		ihs += 4, ++iv;
	}
	
	// Not going to consider inject in GPU implementation at all.
	// Compartments with inject members can be declared as HSolve instead of
	// HSolveCuda.
}

__global__ void forwardEliminateKernel(GpuDataStruct ds) {
	unsigned int ic = 0;
	double *ihs = ds.HS;
	double **iop = ds.operand;
	JunctionStruct *junction;
	
	if ( iop ) {
		for( int x = 0 ; x < 36 ; x++ ) {		//XXX debugging only
			//printf( "%p ", *(iop + x) );
		}
		//printf("\n");
	}
	
	double pivot;
	double division;
	unsigned int index;
	unsigned int rank;
	double *j, *s;
	for ( junction = ds.junction;
	      junction < ds.junction + ds.junctionSize;
	      junction++ )
	{
		index = junction->index;
		rank = junction->rank;
		
		while ( ic < index ) {
			*( ihs + 4 ) -= *( ihs + 1 ) / *ihs * *( ihs + 1 );
			*( ihs + 7 ) -= *( ihs + 1 ) / *ihs * *( ihs + 3 );
			
			++ic, ihs += 4;
		}
		
		pivot = *ihs;
		if ( rank == 1 ) {
			//printf("rank=1; ");
			//printf("ic: %d ", ic);
			//printf("ihs: %p ", ihs);
			//printf("iop: %p ", iop);
			j = *iop;
			s = *(iop + 1);
			
			//printf( "s: %p\n", s );
			
			division    = *( j + 1 ) / pivot;
			*( s )     -= division * *j;
			*( s + 3 ) -= division * *( ihs + 3 );
			
			iop += 3;
		} else if ( rank == 2 ) {
			//printf("rank=2; ");
			//printf("ic: %d ", ic);
			//printf("ihs: %p ", ihs);
			//printf("iop: %p ", iop);
			j = *iop;
			
			s           = *( iop + 1 );
			//printf( "s: %p ", s );
			division    = *( j + 1 ) / pivot;
			*( s )     -= division * *j;
			*( j + 4 ) -= division * *( j + 2 );
			*( s + 3 ) -= division * *( ihs + 3 );
			
			s           = *( iop + 3 );
			//printf( "s: %p\n", s );
			division    = *( j + 3 ) / pivot;
			*( j + 5 ) -= division * *j;
			*( s )     -= division * *( j + 2 );
			*( s + 3 ) -= division * *( ihs + 3 );
			
			iop += 5;
		} else {
			//printf("rank=%d; ", rank);
			//printf("ic: %d ", ic);
			//printf("ihs: %p ", ihs);
			//printf("iop: %p\n", iop);
			double **end = iop + 3 * rank * ( rank + 1 );
			for ( ; iop < end; iop += 3 )
				**iop -= **( iop + 2 ) / pivot * **( iop + 1 );
		}
		
		++ic, ihs += 4;
	}
	
	while ( ic < ds.nCompts - 1 ) {
		*( ihs + 4 ) -= *( ihs + 1 ) / *ihs * *( ihs + 1 );
		*( ihs + 7 ) -= *( ihs + 1 ) / *ihs * *( ihs + 3 );
		
		++ic, ihs += 4;
	}
}

__global__ void backwardSubstituteKernel(GpuDataStruct ds) {
	// We are reverse iterating here, so all pointers are initialized to the
	// ultimate elements of their respective arrays.
	int 	ic		 = ds.nCompts - 1;
	double *ivmid	 = ds.VMid + ic;
	double *iv		 = ds.V + ic;
	double *ihs		 = ds.HS + 4 * ds.nCompts - 1;
	double **iop	 = ds.operand + ds.operandSize - 1;
	double **ibop	 = ds.backOperand + ds.backOperandSize - 1;
	JunctionStruct *junction = ds.junction + ds.junctionSize - 1;
	
	//printf("In gpu: %lf %lf\n", ihs[-3], ihs[0]);

	*ivmid = *ihs / *( ihs - 3 );
	*iv = 2 * *ivmid - *iv;
	--ic, --ivmid, --iv, ihs -= 4;
	
	int index;
	int rank;
	for ( ;
	      ds.junction != NULL && junction >= ds.junction;
	      junction-- )
	{
		index = junction->index;
		rank = junction->rank;
		
		while ( ic > index ) {
			// ivmid was -1, so now it's +1!
			*ivmid = ( *ihs - *( ihs - 2 ) * *( ivmid + 1 ) ) / *( ihs - 3 );
			*iv = 2 * *ivmid - *iv;
			
			--ic, --ivmid, --iv, ihs -= 4;
		}
		
		if ( rank == 1 ) {
			*ivmid = ( *ihs - **iop * **( iop - 2 ) ) / *( ihs - 3 );
			
			iop -= 3;
		} else if ( rank == 2 ) {
			double *v0 = *( iop );
			double *v1 = *( iop - 2 );
			double *j  = *( iop - 4 );
			
			*ivmid = ( *ihs
			           - *v0 * *( j + 2 )	// j was a vdIterator in forward!
			           - *v1 * *j			// so + remains +!!
			         ) / *( ihs - 3 );
			
			iop -= 5;
		} else {
			*ivmid = *ihs;
			for ( int i = 0; i < rank; ++i ) {
				*ivmid -= **ibop * **( ibop - 1 );
				ibop -= 2;
			}
			*ivmid /= *( ihs - 3 );
			
			iop -= 3 * rank * ( rank + 1 );
		}
		
		*iv = 2 * *ivmid - *iv;
		--ic, --ivmid, --iv, ihs -= 4;
	}
	
	while ( ic >= 0 ) {
		// The ivmid was -1, so now it becomes +1!
		*ivmid = ( *ihs - *( ihs - 2 ) * *( ivmid + 1 ) ) / *( ihs - 3 );
		*iv = 2 * *ivmid - *iv;
		
		--ic, --ivmid, --iv, ihs -= 4;
	}

	//printf("V in gpu: %lf %lf\n", ds.VMid[0], ds.V[0]);
}

__device__ void findRow(GpuLookupTable table, double value, GpuLookupRow &row)
{
	if ( value < table.min )
		value = table.min;
	else if ( value > table.max )
		value = table.max;

	double div = ( value - table.min ) / table.dx;
	unsigned int integer = ( unsigned int )( div );

	row.fraction = div - integer;
	row.row = table.table + integer * table.nColumns;
}

__device__ void lookupTable(GpuLookupTable table, GpuLookupColumn column,
							GpuLookupRow row, double &C1, double &C2)
{
	double a, b;
	double *ap, *bp;

	ap = row.row + column.column;

	bp = ap + table.nColumns;

	a = *ap;
	b = *bp;
	C1 = a + ( b - a ) * row.fraction;

	a = *( ap + 1 );
	b = *( bp + 1 );
	C2 = a + ( b - a ) * row.fraction;
}

__global__ void advanceChannelsKernel(GpuDataStruct ds, double dt)
{
	double			 *iv;
	double			 *istate		 = ds.state;
	int				 *ichannelcount	 = ds.channelCount;
	ChannelStruct	 *ichan			 = ds.channel;
	ChannelStruct	 *chanBoundary;
	unsigned int	 *icacount		 = ds.caCount;
	double			 *ica			 = ds.ca;
	double			 *caBoundary;
	GpuLookupColumn	 *icolumn		 = ds.column;
	GpuLookupRow	 *icarowcompt;
	GpuLookupRow	 **icarow		 = ds.caRow;

	GpuLookupRow vRow;
	double C1, C2;
	for ( iv = ds.V ; iv != ds.V + ds.nCompts ; ++iv ) {
		findRow( ds.vTable, *iv, vRow );
		icarowcompt = ds.caRowCompt;
		caBoundary = ica + *icacount;
		for ( ; ica < caBoundary; ++ica, ++icarowcompt )
			// Implicitly assumes that we are not going out of bounds for
			// caRowCompt! => The size of caRowCompt must be the max number of
			// calcium pools out of all compartments.
			findRow( ds.caTable, *ica, *icarowcompt );

		/*
		 * Optimize by moving "if ( instant )" outside the loop, because it is
		 * rarely used. May also be able to avoid "if ( power )".
		 *
		 * Or not: excellent branch predictors these days.
		 *
		 * Will be nice to test these optimizations.
		 */
		chanBoundary = ichan + *ichannelcount;
		for ( ; ichan < chanBoundary ; ++ichan ) {
			if ( ichan->Xpower_ > 0.0 ) {
				lookupTable( ds.vTable, *icolumn, vRow, C1, C2 );
				if ( ichan->instant_ & INSTANT_X )
					*istate = C1 / C2;
				else {
					double temp = 1.0 + dt / 2.0 * C2;
					*istate = ( *istate * ( 2.0 - temp ) + dt * C1 ) / temp;
				}
				++icolumn, ++istate;
			}

			if ( ichan->Ypower_ > 0.0 ) {
				lookupTable( ds.vTable, *icolumn, vRow, C1, C2 );
				if ( ichan->instant_ & INSTANT_Y )
					*istate = C1 / C2;
				else {
					double temp = 1.0 + dt / 2.0 * C2;
					*istate = ( *istate * ( 2.0 - temp ) + dt * C1 ) / temp;
				}
				++icolumn, ++istate;
			}

			if ( ichan->Zpower_ > 0.0 ) {
				GpuLookupRow *caRow = *icarow;
				if ( caRow ) {
					lookupTable( ds.caTable, *icolumn, *caRow, C1, C2 );
				} else {
					lookupTable( ds.vTable, *icolumn, vRow, C1, C2 );
				}

				if ( ichan->instant_ & INSTANT_Z )
					*istate = C1 / C2;
				else {
					double temp = 1.0 + dt / 2.0 * C2;
					*istate = ( *istate * ( 2.0 - temp ) + dt * C1 ) / temp;
				}

				++icolumn, ++istate, ++icarow;
			}
		}

		++ichannelcount, ++icacount;
	}
}

__global__ void calculateChannelCurrentsKernel(GpuDataStruct ds)
{
	ChannelStruct *ichan;
	CurrentStruct *icurrent = ds.current;

	if ( ds.stateSize != 0 ) {
		double *istate = ds.state;

		for ( ichan = ds.channel ; ichan != ds.channel + ds.nChannels ;
			  ++ichan, ++icurrent )
		{
			// Stuff inside ichan->process
			double fraction = 1;
			// No complications for taking power
			if ( ichan->Xpower_ > 0.0 )
				fraction *= pow( *( istate++ ), ichan->Xpower_ );
			if ( ichan->Ypower_ > 0.0 )
				fraction *= pow( *( istate++ ), ichan->Ypower_ );
			if ( ichan->Zpower_ > 0.0 )
				fraction *= pow( *( istate++ ), ichan->Zpower_ );
			icurrent->Gk = ichan->Gbar_ * fraction;
		}
	}
}

__global__ void advanceCalciumKernel(GpuDataStruct ds)
{
	double **icatarget = ds.caTarget;
	double *ivmid = ds.VMid;
	CurrentStruct *icurrent = ds.current;
	CurrentStruct **iboundary = ds.currentBoundary;

	// Reset caActivation to zero
	memset( ds.caActivation, 0.0, ds.nCaPools );

	// caAdvance is just taken to be 1. The 0 case is not implemented.

	// There are as many current boundaries as there are compartments.
	for ( ; iboundary != ds.currentBoundary + ds.nCompts ; ++iboundary ) {
		for ( ; icurrent < *iboundary ; ++icurrent ) {
			if ( *icatarget )
				**icatarget += icurrent->Gk * ( icurrent->Ek - *ivmid );
			++icatarget;
		}
		++ivmid;
	}

	CaConcStruct *icaconc;
	double *icaactivation = ds.caActivation;
	double *ica = ds.ca;
	for( icaconc = ds.caConc; icaconc != ds.caConc + ds.nCaPools; ++icaconc ) {
		// CaConcStruct::process has been inserted here, fully expanded
		icaconc->c_ = 	icaconc->factor1_ * icaconc->c_
					  + icaconc->factor2_ * *icaactivation;
		double ca = icaconc->CaBasal_ + icaconc->c_;
		if ( icaconc->ceiling_ > 0 && ca > icaconc->ceiling_ ) {
			ca = icaconc->ceiling_;
			icaconc->c_ = ca - icaconc->CaBasal_;	// CaConcStruct::setCa
		}
		if ( ca < icaconc->floor_ ) {
			ca = icaconc->floor_;
			icaconc->c_ = ca - icaconc->CaBasal_;	// CaConcStruct::setCa
		}
		*ica = ca;
		++ica, ++icaactivation;
	}
}
