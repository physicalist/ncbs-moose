
#include <hip/hip_runtime.h>
#ifdef DO_UNIT_TESTS

#include "testGpuKernels.h"
#include "GpuInterface.h"
#include "GpuKernels.h"

#include <limits>
/**
 * Check 2 floating-point numbers for "equality".
 * Algorithm (from Knuth) 'a' and 'b' are close if:
 *      | ( a - b ) / a | < e AND | ( a - b ) / b | < e
 * where 'e' is a small number.
 *
 * In this function, 'e' is computed as:
 *      e = tolerance * machine-epsilon
 */
template< class T >
bool isClose( T a, T b, T tolerance )
{
	T epsilon = std::numeric_limits< T >::epsilon();

	if ( a == b )
		return true;

	if ( a == 0 || b == 0 )
		return ( fabs( a - b ) < tolerance * epsilon );

	return (    fabs( ( a - b ) / a ) < tolerance * epsilon
			 && fabs( ( a - b ) / b ) < tolerance * epsilon );
}

const double tolerance = 1.0;

void testHSonly()
{
	// Fabricate a GpuDataStruct for testing purposes and check whether or
	// not forward elimination and backward substitution occur properly.

	GpuDataStruct d;

	// First create a test matrix.
	const int n = 4;				// Start with a 4x4 matrix
	double testHS[] = {	2, 1, 0, 4,
						3, 1, 0, 10,
						4, 1, 0, 18,
						5, 0, 0, 23 };	// Equivalent of HS => tridiagonal
	double V[] = { 0, 0, 0, 0 };
	double VMid[] = { 0, 0, 0, 0 };

	// Allocate memory as required for d.
	cudaMalloc( (void **)&d.HS, 4 * n * sizeof(double) );
	d.HJ = NULL;
	cudaMalloc( (void **)&d.V, n * sizeof(double) );
	cudaMalloc( (void **)&d.VMid, n * sizeof(double) );
	d.HJCopy = NULL;
	d.operand = NULL;		// Not required if there are no junctions!
	d.backOperand = NULL;	// Likewise!
	d.compartment = NULL;	// Not used by fwdElim and bwdSubs.
	d.junction = NULL;		// No junctions for now.

	d.nCompts = n;
	d.HJSize = 0;
	d.operandSize = 0;
	d.backOperandSize = 0;
	d.junctionSize = 0;

	// Copy HS data into d.
	cudaMemcpy( d.HS, testHS, 4 * n * sizeof(double), cudaMemcpyHostToDevice );
	cudaMemcpy( d.V, V, n * sizeof(double), cudaMemcpyHostToDevice );
	cudaMemcpy( d.VMid, VMid, n * sizeof(double), cudaMemcpyHostToDevice );

	// Call forward eliminate and backward substitute.
	dim3 numBlocks(1);
	dim3 numThreads(1);
	cudaStream_t stream;
	cudaStreamCreate( &stream );
	forwardEliminateKernel<<< numBlocks, numThreads >>>(d);

	// Check the matrix after forward eliminate
	cudaMemcpy( testHS, d.HS, 4 * n * sizeof(double), cudaMemcpyDeviceToHost );
	//std::cout << "HS after forward eliminate" << std::endl;
	int i;
	for( i = 0 ; i < 4 * n ; i++ ) {
		//std::cout << testHS[i] << " ";
		if( i % 4 == 3 ) {
			//std::cout << std::endl;
		}
	}

	backwardSubstituteKernel<<< numBlocks, numThreads >>>(d);

	// Copy results back into CPU
	cudaMemcpy( V, d.V, n * sizeof(double), cudaMemcpyDeviceToHost );
	cudaMemcpy( VMid, d.VMid, n * sizeof(double), cudaMemcpyDeviceToHost );

	//std::cout << "V" << std::endl;
	for( i = 0 ; i < n ; i++ ) {
		//std::cout << V[i] << " ";
	}

	//std::cout << std::endl << "VMid" << std::endl;
	for( i = 0 ; i < n ; i++ ) {
		//std::cout << VMid[i] << " ";
	}

	ASSERT( isClose( VMid[0], 1.0, tolerance ), "Error: testHSonly, VMid[0]");
	ASSERT( isClose( VMid[1], 2.0, tolerance ), "Error: testHSonly, VMid[1]");
	ASSERT( isClose( VMid[2], 3.0, tolerance ), "Error: testHSonly, VMid[2]");
	ASSERT( isClose( VMid[3], 4.0, tolerance ), "Error: testHSonly, VMid[3]");

	std::cout << ".";
}

void testYcompt()
{
	// Fabricate a GpuDataStruct for testing purposes and check whether or
	// not forward elimination and backward substitution occur properly.

	// The network in question is a simple Y-network with three compartments.

	GpuDataStruct d;

	// First create a test matrix along with associated data structures.
	const int n = 3;				// Start with a 3x3 matrix
	double testHS[] = {	1, 0, 0, -4,
						2, 0, 0, 0,
						3, 0, 0, 6 };	// Equivalent of HS => tridiagonal
	double testHJ[] = { -1, -1, -1, -1, -1, -1 };
	double **testOperand = new double*[8];
	JunctionStruct testJunction[2] = { JunctionStruct::JunctionStruct(0, 2),
									   JunctionStruct::JunctionStruct(1, 1) };
	double V[] = { 0, 0, 0 };
	double VMid[] = { 0, 0, 0 };

	// Allocate memory as required for d.
	cudaMalloc( (void **)&d.HS, 4 * n * sizeof(double) );
	cudaMalloc( (void **)&d.HJ, n * (n-1) * sizeof(double) );
	cudaMalloc( (void **)&d.V, n * sizeof(double) );
	cudaMalloc( (void **)&d.VMid, n * sizeof(double) );
	cudaMalloc( (void **)&d.HJCopy, n * (n-1) * sizeof(double) );
	cudaMalloc( (void **)&d.operand, 8 * sizeof(double *) ); // XXX n=3 only!
	d.backOperand = NULL;	// Not reqd if junctions have rank < 3
	d.compartment = NULL;	// Not needed unless updateMatrix is being called
	cudaMalloc( (void **)&d.junction, 2 * sizeof(JunctionStruct) );
									// XXX For this network only!

	d.nCompts = n;
	d.HJSize = n * (n-1);
	d.operandSize = 8;		// XXX
	d.backOperandSize = 0;
	d.junctionSize = 2;		// XXX

	// Set up operands
	testOperand[0] = d.HJ;
	testOperand[1] = d.HS + 4;
	testOperand[2] = d.VMid + 1;
	testOperand[3] = d.HS + 8;
	testOperand[4] = d.VMid + 2;
	testOperand[5] = d.HJ + 4;
	testOperand[6] = d.HS + 8;
	testOperand[7] = d.VMid + 2;

	//std::cout << "Addresses:" << std::endl;
	//std::cout << d.HS << " " << testOperand[6] << " " << d.HJ << std::endl;
	//std::cout << "Operand address: " << d.operand << std::endl;
	//std::cout << "V and VMid addresses: " << d.V << " " << d.VMid << std::endl;

	// Copy data into device.
	cudaMemcpy( d.HS, testHS, 4 * n * sizeof(double), cudaMemcpyHostToDevice );
	cudaMemcpy( d.HJ, testHJ, n * (n-1) * sizeof(double),
				cudaMemcpyHostToDevice );
	cudaMemcpy( d.operand, testOperand, 8 * sizeof(double *),
				cudaMemcpyHostToDevice );
	cudaMemcpy( d.junction, testJunction, 2 * sizeof(JunctionStruct),
				cudaMemcpyHostToDevice );
	cudaMemcpy( d.V, V, n * sizeof(double), cudaMemcpyHostToDevice );
	cudaMemcpy( d.VMid, VMid, n * sizeof(double), cudaMemcpyHostToDevice );

	cudaDeviceSynchronize();

	// Call forward eliminate and backward substitute.
	dim3 numBlocks(1);
	dim3 numThreads(1);
	cudaStream_t stream;
	cudaStreamCreate( &stream );
	forwardEliminateKernel<<< numBlocks, numThreads, 0, stream >>>(d);

	// Check the matrix after forward eliminate
	cudaMemcpy( testHS, d.HS, 4 * n * sizeof(double), cudaMemcpyDeviceToHost );
	//std::cout << "HS after forward eliminate" << std::endl;
	int i;
	for( i = 0 ; i < 4 * n ; i++ ) {
		//std::cout << testHS[i] << " ";
		if( i % 4 == 3 ) {
			//std::cout << std::endl;
		}
	}
	cudaMemcpy( testHJ, d.HJ, n * (n-1) * sizeof(double),
				cudaMemcpyDeviceToHost );
	//std::cout << "HJ after forward eliminate" << std::endl;
	for( i = 0 ; i < n * (n-1) ; i++ ) {
		//std::cout << testHJ[i] << " ";
	}
	//std::cout << endl;

	backwardSubstituteKernel<<< numBlocks, numThreads, 0, stream >>>(d);

	// Copy results back into CPU
	cudaMemcpy( V, d.V, n * sizeof(double), cudaMemcpyDeviceToHost );
	cudaMemcpy( VMid, d.VMid, n * sizeof(double), cudaMemcpyDeviceToHost );

	//std::cout << "V" << std::endl;
	for( i = 0 ; i < n ; i++ ) {
		//std::cout << V[i] << " ";
	}

	//std::cout << std::endl << "VMid" << std::endl;
	for( i = 0 ; i < n ; i++ ) {
		//std::cout << VMid[i] << " ";
	}

	ASSERT( isClose( VMid[0], 1.0, tolerance ), "Error: testYcompt, VMid[0]");
	ASSERT( isClose( VMid[1], 2.0, tolerance ), "Error: testYcompt, VMid[1]");
	ASSERT( isClose( VMid[2], 3.0, tolerance ), "Error: testYcompt, VMid[2]");

	std::cout << ".";
}

void testRank3()
{
	// Fabricate a GpuDataStruct for testing purposes and check whether or
	// not forward elimination and backward substitution occur properly.

	// The network in question is a simple Y-network with three compartments.

	GpuDataStruct d;

	// First create a test matrix along with associated data structures.
	const int n = 4;				// Start with a 3x3 matrix
	double testHS[] = {	1, 0, 0, -8,
						2, 0, 0, -4,
						3, 0, 0, 2,
						4, 0, 0, 10 };	// Equivalent of HS => tridiagonal
	double testHJ[] = { -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 };
	JunctionStruct testJunction[3] = { JunctionStruct::JunctionStruct(0, 3),
									   JunctionStruct::JunctionStruct(1, 2),
									   JunctionStruct::JunctionStruct(2, 1) };
	double V[] = { 0, 0, 0, 0 };
	double VMid[] = { 0, 0, 0, 0 };

	// Allocate memory as required for d.
	cudaMalloc( (void **)&d.HS, 4 * n * sizeof(double) );
	cudaMalloc( (void **)&d.HJ, n * (n-1) * sizeof(double) );
	cudaMalloc( (void **)&d.V, n * sizeof(double) );
	cudaMalloc( (void **)&d.VMid, n * sizeof(double) );
	cudaMalloc( (void **)&d.HJCopy, n * (n-1) * sizeof(double) );
	cudaMalloc( (void **)&d.operand, 44 * sizeof(double *) ); // XXX n=4 only!
	cudaMalloc( (void **)&d.backOperand, 6 * sizeof(double *) ); // XXX
	d.compartment = NULL;	// Not needed unless updateMatrix is being called
	cudaMalloc( (void **)&d.junction, 3 * sizeof(JunctionStruct) );
									// XXX For this network only!

	d.nCompts = n;
	d.HJSize = n * (n-1);
	d.operandSize = 44;		// XXX
	d.backOperandSize = 6;
	d.junctionSize = 3;		// XXX

	// Set up operands
	double *testOperand[44] = {
							// junction[0]
								d.HS + 4,	// 0
								d.HJ,		// 1
								d.HJ + 1,	// 2
								d.HS + 7,	// 3
								d.HS + 3,	// 4
								d.HJ + 1,	// 5
								d.HS + 8,	// 6
								d.HJ + 2,	// 7
								d.HJ + 3,	// 8
								d.HS + 11,	// 9
								d.HS + 3,	// 10
								d.HJ + 3,	// 11
								d.HS + 12,	// 12
								d.HJ + 4,	// 13
								d.HJ + 5,	// 14
								d.HS + 15,	// 15
								d.HS + 3,	// 16
								d.HJ + 5,	// 17
								d.HJ + 6,	// 18
								d.HJ + 2,	// 19
								d.HJ + 1,	// 20
								d.HJ + 8,	// 21
								d.HJ + 4,	// 22
								d.HJ + 1,	// 23
								d.HJ + 10,	// 24
								d.HJ + 4,	// 25
								d.HJ + 3,	// 26
								d.HJ + 7,	// 27
								d.HJ,		// 28
								d.HJ + 3,	// 29
								d.HJ + 9,	// 30
								d.HJ,		// 31
								d.HJ + 5,	// 32
								d.HJ + 11,	// 33
								d.HJ + 2,	// 34
								d.HJ + 5,	// 35
							// junction[1]
								d.HJ + 6,	// 36
								d.HS + 8,	// 37
								d.VMid + 2,	// 38
								d.HS + 12,	// 39
								d.VMid + 3,	// 40
							// junction[2]
								d.HJ + 10,	// 41
								d.HS + 12,	// 42
								d.VMid + 3,	// 43
							  };

	double *testBackOperand[6] = {
									d.HJ,		// 0
									d.VMid + 1,	// 1
									d.HJ + 2,	// 2
									d.VMid + 2,	// 3
									d.HJ + 4,	// 4
									d.VMid + 3	// 5
								 };

	//std::cout << "Addresses:" << std::endl;
	//std::cout << "HS: " << d.HS << " " << "HJ: " << d.HJ << std::endl;
	//std::cout << "Operand address: " << d.operand << std::endl;
	//std::cout << "V and VMid addresses: " << d.V << " " << d.VMid << std::endl;

	// Copy data into device.
	cudaMemcpy( d.HS, testHS, 4 * n * sizeof(double), cudaMemcpyHostToDevice );
	cudaMemcpy( d.HJ, testHJ, n * (n-1) * sizeof(double),
				cudaMemcpyHostToDevice );
	cudaMemcpy( d.operand, testOperand, 44 * sizeof(double *),
				cudaMemcpyHostToDevice );
	cudaMemcpy( d.backOperand, testBackOperand, 6 * sizeof(double *),
				cudaMemcpyHostToDevice );
	cudaMemcpy( d.junction, testJunction, 3 * sizeof(JunctionStruct),
				cudaMemcpyHostToDevice );
	cudaMemcpy( d.V, V, n * sizeof(double), cudaMemcpyHostToDevice );
	cudaMemcpy( d.VMid, VMid, n * sizeof(double), cudaMemcpyHostToDevice );

	// Call forward eliminate and backward substitute.
	dim3 numBlocks(1);
	dim3 numThreads(1);
	cudaStream_t stream;
	cudaStreamCreate( &stream );
	forwardEliminateKernel<<< numBlocks, numThreads, 0, stream >>>(d);

	// Check the matrix after forward eliminate
	cudaMemcpy( testHS, d.HS, 4 * n * sizeof(double), cudaMemcpyDeviceToHost );
	//std::cout << "HS after forward eliminate" << std::endl;
	int i;
	for( i = 0 ; i < 4 * n ; i++ ) {
		//std::cout << testHS[i] << " ";
		if( i % 4 == 3 ) {
			//std::cout << std::endl;
		}
	}
	cudaMemcpy( testHJ, d.HJ, n * (n-1) * sizeof(double),
				cudaMemcpyDeviceToHost );
	//std::cout << "HJ after forward eliminate" << std::endl;
	for( i = 0 ; i < n * (n-1) ; i++ ) {
		//std::cout << testHJ[i] << " ";
	}
	//std::cout << endl;

	backwardSubstituteKernel<<< numBlocks, numThreads, 0, stream >>>(d);

	// Copy results back into CPU
	cudaMemcpy( V, d.V, n * sizeof(double), cudaMemcpyDeviceToHost );
	cudaMemcpy( VMid, d.VMid, n * sizeof(double), cudaMemcpyDeviceToHost );

	//std::cout << "V" << std::endl;
	for( i = 0 ; i < n ; i++ ) {
		//std::cout << V[i] << " ";
	}

	//std::cout << std::endl << "VMid" << std::endl;
	for( i = 0 ; i < n ; i++ ) {
		//std::cout << VMid[i] << " ";
	}

	ASSERT( isClose( VMid[0], 1.0, tolerance ), "Error: testRank3, VMid[0]");
	ASSERT( isClose( VMid[1], 2.0, tolerance ), "Error: testRank3, VMid[1]");
	ASSERT( isClose( VMid[2], 3.0, tolerance ), "Error: testRank3, VMid[2]");
	ASSERT( isClose( VMid[3], 4.0, tolerance ), "Error: testRank3, VMid[3]");

	std::cout << ".";
}

void testGpuKernels()
{
	std::cout << "Testing GpuKernels: " << std::flush;
	testHSonly();
	testYcompt();
	testRank3();
	std::cout << std::endl;
}

#endif // DO_UNIT_TESTS
